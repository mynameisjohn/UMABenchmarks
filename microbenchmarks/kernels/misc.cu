#include "hip/hip_runtime.h"
#include <cuda_occupancy.h>
#include <hip/hip_runtime_api.h>

#include "microbenchmarks.h"

#include "util.h"

__global__
void inc( float * data, int N )
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if ( idx < N )
		data[idx]++;
}

LaunchParams GetBestOccupancy( void * kernel, int N )
{
	LaunchParams ret = { 0, 0 };
	uint32_t minGridSize( 0 );
	// find best occupancy stuff (not working on windows)
#ifndef _WIN32
	hipModuleOccupancyMaxPotentialBlockSize( &minGridSize.z, &ret.numThreads, (hipFunction_t) kernel, 0, 0, 0 );
#else
	ret.numThreads = 1024;
#endif

	// Pick a sensible block number
	ret.numBlocks = ( N + ret.numThreads - 1 ) / ret.numThreads;

	return ret;
}

std::string TestFunc::GetName() const
{
	return m_StrName;
}

ScopedCuProfiler::ScopedCuProfiler()
{
	hipProfilerStart();
}

ScopedCuProfiler::~ScopedCuProfiler()
{
	hipProfilerStop();
}