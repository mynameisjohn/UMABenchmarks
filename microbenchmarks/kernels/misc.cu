#include "hip/hip_runtime.h"
#include <cuda_occupancy.h>
#include <hip/hip_runtime_api.h>

#include "microbenchmarks.h"

#include "util.h"

__global__
void inc( float * data, int N )
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if ( idx < N )
		data[idx]++;
}

LaunchParams GetBestOccupancy( void * kernel, int N )
{
	LaunchParams ret = { 0, 0 };
	int minGridSize( 0 );
	// find best occupancy stuff (not working on windows)
	ret.numThreads = 1024;

	// Pick a sensible block number
	ret.numBlocks = ( N + ret.numThreads - 1 ) / ret.numThreads;

	return ret;
}

std::string TestFunc::GetName() const
{
	return m_StrName;
}

ScopedCuProfiler::ScopedCuProfiler()
{
	hipProfilerStart();
}

ScopedCuProfiler::~ScopedCuProfiler()
{
	hipProfilerStop();
}
