#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>


#include "../util.h"
#include "microbenchmarks.h"

const float minRes( 0.1f );
inline unsigned int sqrtToInt( int i )
{
	return (unsigned int) ( sqrt( i + 0.1 ) );
}

__inline__ __host__ __device__
uint32_t get2Didx( uint32_t x, uint32_t y, uint32_t N )
{
	return x + y * N;
}

__global__
void gsRelax_Laplacian2D_even( float * in, float * out, uint32_t N )
{
	uint32_t idx_X = 2 * ( threadIdx.x + blockDim.x*blockIdx.x );
	uint32_t idx_Y = threadIdx.y + blockDim.y*blockIdx.y;

	if ( idx_X > 0 && idx_X < N - 1 && idx_Y > 0 && idx_Y < N - 1 )
	{
		uint32_t idx = get2Didx( idx_X, idx_Y, N );
		uint32_t idx_x1 = get2Didx( idx_X - 1, idx_Y, N );
		uint32_t idx_x2 = get2Didx( idx_X + 1, idx_Y, N );
		uint32_t idx_y1 = get2Didx( idx_X, idx_Y - 1, N );
		uint32_t idx_y2 = get2Didx( idx_X, idx_Y + 1, N );

		float sum =
			in[idx_x1] +
			in[idx_x2] +
			in[idx_y1] +
			in[idx_y2];

		out[idx] = 0.25f * sum;
	}
}

__global__
void gsRelax_Laplacian2D_odd( float * in, float * out, uint32_t N )
{
	uint32_t idx_X = 2 * ( threadIdx.x + blockDim.x*blockIdx.x ) + 1;
	uint32_t idx_Y = threadIdx.y + blockDim.y*blockIdx.y;

	if ( idx_X > 0 && idx_X < N - 1 && idx_Y > 0 && idx_Y < N - 1 )
	{
		uint32_t idx = get2Didx( idx_X, idx_Y, N );
		uint32_t idx_x1 = get2Didx( idx_X - 1, idx_Y, N );
		uint32_t idx_x2 = get2Didx( idx_X + 1, idx_Y, N );
		uint32_t idx_y1 = get2Didx( idx_X, idx_Y - 1, N );
		uint32_t idx_y2 = get2Didx( idx_X, idx_Y + 1, N );

		float sum =
			in[idx_x1] +
			in[idx_x2] +
			in[idx_y1] +
			in[idx_y2];

		out[idx] = 0.25f * sum;
	}
}

__global__
void gsRelax_Laplacian1D_even( float * in, float * out, uint32_t N)
{
	uint32_t idx = 2 * ( threadIdx.x + blockDim.x*blockIdx.x );

	if ( idx > 0 && idx < N - 1 )
	{
		float sum = in[idx - 1] + in[idx + 1];
		out[idx] = 0.5f * sum;
	}
}

__global__
void gsRelax_Laplacian1D_odd( float * in, float * out, uint32_t N )
{
	uint32_t idx = 2 * ( threadIdx.x + blockDim.x*blockIdx.x ) + 1;

	if ( idx > 0 && idx < N - 1 )
	{
		float sum = in[idx - 1] + in[idx + 1];
		out[idx] = 0.5f * sum;
	}
}


inline float getResidueSq( float * in, float * out, uint32_t N )
{
	float r( 0 );
	for ( uint32_t i = 0; i<N; i++ )
		r += pow( out[i] - in[i], 2 );
	return r;
}

void makeData( float * data, uint32_t N )
{
	srand( 1 );//time(0));
	for ( uint32_t i = 0; i<N; i++ )
		data[i] = (float) rand() / (float) RAND_MAX;
}


float RelaxFunc::runUMA( uint32_t N, uint32_t dim, uint32_t nIt )
{
	// Create timing objects, do not start
	float timeTaken( 0 );
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );

	//Just a stupid pad
	dim = ( dim % 2 ? 1 : 2 );
	N = ( dim == 1 ? N : N*N );
	uint32_t size = sizeof( float )*N;
	float * d_Data_A( 0 ), *d_Data_B( 0 );
	float res( 0 );
	hipMallocManaged( (void **) &d_Data_A, size );
	hipMallocManaged( (void **) &d_Data_B, size );

	// Get max occupancy values
	LaunchParams occ = GetBestOccupancy( dim == 2 ? gsRelax_Laplacian1D_even : gsRelax_Laplacian2D_even, N );

	// Start timing
	hipEventRecord( start );

	// Create random data
	makeData( d_Data_A, N );
	makeData( d_Data_B, N );

	//Repetitive, I know, but I didn't want to introduce overhead during iteration
	if ( dim == 1 )
	{
		//CudaStopWatch CSW( "UMA" );
		//int nT( 1024 ), nB( ( N / 1024 ) / 2 + 1 );
		for ( int i = 0; i < nIt; i++ )
		{
			gsRelax_Laplacian1D_even << <occ.numBlocks, occ.numThreads >> >( d_Data_A, d_Data_B, N );
			gsRelax_Laplacian1D_odd << < occ.numBlocks, occ.numThreads >> >( d_Data_A, d_Data_B, N );
			hipDeviceSynchronize();
			res = sqrt( getResidueSq( d_Data_A, d_Data_B, N ) );

			swap( d_Data_A, d_Data_B );
		}
	}
	else if ( dim == 2 )
	{
		// Right now it's assumed there's a nice sqrt of numThreads (usually 1024 ==> 32x32)
		uint32_t len = sqrt( N );
		uint3 numThreads = make_uint3( sqrtToInt( occ.numThreads ), sqrtToInt( occ.numThreads ), 0 );
		uint3 numBlocks = make_uint3( len / occ.numThreads, 1, 0 ); // not sure about this
		for ( int i = 0; i < nIt; i++ )
		{
			gsRelax_Laplacian2D_even << <numBlocks, numThreads >> >( d_Data_A, d_Data_B, len );
			gsRelax_Laplacian2D_odd << <numBlocks, numThreads >> >( d_Data_A, d_Data_B, len );
			hipDeviceSynchronize();
			res = sqrt( getResidueSq( d_Data_A, d_Data_B, N ) );

			swap( d_Data_A, d_Data_B );
		}
	}

	// Stop timing
	hipEventRecord( stop );
	hipEventSynchronize( stop );

	// Get elapsed time
	hipEventElapsedTime( &timeTaken, start, stop );

	// Free data
	hipFree( d_Data_A );
	hipFree( d_Data_B );

	return timeTaken;
}

float RelaxFunc::runHD( uint32_t N, uint32_t dim, uint32_t nIt )
{
	// Create timing objects, do not start
	float timeTaken( 0 );
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );

	//Just a stupid pad
	dim = ( dim % 2 ? 1 : 2 );
	N = ( dim == 1 ? N : N*N );
	uint32_t size = sizeof( float )*N;
	float * h_Data_A( 0 ), *d_Data_A( 0 ), *h_Data_B( 0 ), *d_Data_B( 0 );
	float res( 0 );
	h_Data_A = (float *) malloc( size );
	h_Data_B = (float *) malloc( size );
	hipMalloc( (void **) &d_Data_A, size );
	hipMalloc( (void **) &d_Data_B, size );

	// Get max occupancy values
	LaunchParams occ = GetBestOccupancy( dim == 2 ? gsRelax_Laplacian1D_even : gsRelax_Laplacian2D_even, N );

	// Start timing
	hipEventRecord( start );

	makeData( h_Data_A, N );
	makeData( h_Data_B, N );

	//Repetitive, I know, but I didn't want to introduce overhead during iteration
	if ( dim == 1 )
	{
		//CudaStopWatch CSW( "UMA" );
		//int nT( 1024 ), nB( ( N / 1024 ) / 2 + 1 );
		for ( int i = 0; i < nIt; i++ )
		{
			hipMemcpy( d_Data_A, h_Data_A, size, hipMemcpyHostToDevice );
			hipMemcpy( d_Data_B, h_Data_B, size, hipMemcpyHostToDevice );
			gsRelax_Laplacian1D_even << < occ.numBlocks, occ.numThreads >> >( d_Data_A, d_Data_B, N );
			gsRelax_Laplacian1D_odd << < occ.numBlocks, occ.numThreads >> >( d_Data_A, d_Data_B, N );
			hipMemcpy( h_Data_A, d_Data_A, size, hipMemcpyDeviceToHost );
			hipMemcpy( h_Data_B, d_Data_B, size, hipMemcpyDeviceToHost );
			res = sqrt( getResidueSq( h_Data_A, h_Data_B, N ) );

			swap( h_Data_A, h_Data_B );
			swap( d_Data_A, d_Data_B );
		}
	}
	else if ( dim == 2 )
	{
		// Right now it's assumed there's a nice sqrt of numThreads (usually 1024 ==> 32x32)
		uint32_t len = sqrt( N );
		uint3 numThreads = make_uint3( sqrtToInt( occ.numThreads ), sqrtToInt( occ.numThreads ), 0 );
		uint3 numBlocks = make_uint3( len / occ.numThreads, 1, 0 ); // not sure about this
		for ( int i = 0; i < nIt; i++ )
		{
			hipMemcpy( d_Data_A, h_Data_A, size, hipMemcpyHostToDevice );
			hipMemcpy( d_Data_B, h_Data_B, size, hipMemcpyHostToDevice );
			gsRelax_Laplacian2D_even << <numBlocks, numThreads >> >( d_Data_A, d_Data_B, len );
			gsRelax_Laplacian2D_odd << <numBlocks, numThreads >> >( d_Data_A, d_Data_B, len );
			hipMemcpy( h_Data_A, d_Data_A, size, hipMemcpyDeviceToHost );
			hipMemcpy( h_Data_B, d_Data_B, size, hipMemcpyDeviceToHost );
			res = sqrt( getResidueSq( h_Data_A, h_Data_B, N ) );

			swap( h_Data_A, h_Data_B );
			swap( d_Data_A, d_Data_B );
		}
	}

	// Stop timing
	hipEventRecord( stop );
	hipEventSynchronize( stop );

	// Get elapsed time
	hipEventElapsedTime( &timeTaken, start, stop );

	// Free data
	free( h_Data_A );
	free( h_Data_B );
	hipFree( d_Data_A );
	hipFree( d_Data_B );

	return timeTaken;
}
//
//// I'm keeping this around as a generic way of getting occupancy optima
//__global__ void MyKernel( int *array, int arrayCount )
//{
//	int idx = threadIdx.x + blockIdx.x * blockDim.x;
//	if ( idx < arrayCount )
//	{
//		array[idx] *= array[idx];
//	}
//}
//
//int runRelax( int argc, char ** argv )
//{
//	// Get problem size, # iterations, dimension
//	int N = atoi( argv[2] );
//	int dim = atoi( argv[3] );
//	int nIt = atoi( argv[4] );
//
//	if ( N < 0 || dim < 0 || nIt < 0 )
//	{
//		printf( "Error! Invalid arguments passed:\n" );
//		for ( int i = 0; i < argc; i++ )
//			printf( "%s\n", argv[i] );
//		return EXIT_FAILURE;
//	}
//
//
//	// See if profiling or benchmarking
//	std::string type = argv[1];
//	if ( type == "profile")
//	{
//		std::string pattern = argv[5];
//		hipProfilerStart();
//		if ( pattern == "UMA" )
//		{
//			gsRelax_UMA( N, dim, nIt );
//		}
//		else if ( pattern == "HD" )
//		{
//			gsRelax_HD( N, dim, nIt );
//		}
//		hipProfilerStop();
//
//		return EXIT_SUCCESS;
//	}
//	// We need number of times run for benchmarking
//	else if ( type == "benchmark" && argc >= 6)
//	{
//		int testCount = atoi( argv[5] );
//		
//		// Do both UMA and Host-Device code
//		// Create a cuda event, start timing, stop, write to file
//		float umaSum( 0 ), hdSum( 0 );
//		for ( int i = 0; i < testCount; i++ )
//		{
//			hdSum += gsRelax_HD( N, dim, nIt );
//			umaSum += gsRelax_UMA( N, dim, nIt );
//		}
//
//		// Find average runtime
//		hdSum /= float( testCount );
//		umaSum /= float( testCount );
//
//		// Print to file based on prob size
//		std::string fileName = "gsRelax_";
//		fileName.append( argv[2] ).append(".txt");
//		FILE * fp = fopen( fileName.c_str(), "w" );
//		if ( !fp )
//		{
//			printf( "Error opening file %s! closing...\n", fileName.c_str() );
//			return EXIT_FAILURE;
//		}
//
//		fprintf( fp, "%f\t%f", hdSum, umaSum );
//		fclose( fp );
//
//		return EXIT_SUCCESS;
//	}
//
//	return EXIT_FAILURE;
//}