#include "hip/hip_runtime.h"
#include "util.h"
#include "microbenchmarks.h"

float AGACFunc::runUMA( uint32_t N, uint32_t dim, uint32_t nIt )
{
	// Create timing objects, do not start
	CpuTimer T;

	// Allocate data
	size_t size = sizeof( float ) * N;
	float *d_Data( 0 );
	hipMallocManaged( (void **) &d_Data, size );

	// Get max occupancy values
	LaunchParams occ = GetBestOccupancy( (void *)inc, N );

	// Start timing
	T.Start();

	// Make random data between 0 and N
	makeData( d_Data, N );

	// Increment everything on CPU and GPU
	for ( int i = 0; i < nIt; i++ )
	{
		inc << < occ.numBlocks, occ.numThreads >> >( d_Data, N );
		hipDeviceSynchronize();
		incData( d_Data, N );
	}

	// Get elapsed time
	hipDeviceSynchronize();
	float timeTaken = T.Elapsed();

	// Free
	hipFree( d_Data );

	return timeTaken;
}

float AGACFunc::runHD( uint32_t N, uint32_t dim, uint32_t nIt )
{
	// Create timing objects, do not start
	CpuTimer T;

	// Allocate data
	size_t size = sizeof( float ) * N;
	float * h_Data( 0 ), *d_Data( 0 );
	h_Data = (float *) malloc( size );
	hipMalloc( (void **) &d_Data, size );

	// Get max occupancy values
	LaunchParams occ = GetBestOccupancy( (void *) inc, N );

	// Start timing
	T.Start();

	// Make random data between 0 and N
	makeData( h_Data, N );
	
	// Increment everything on CPU and GPU
	for ( int i = 0; i < nIt; i++ )
	{
		hipMemcpy( d_Data, h_Data, size, hipMemcpyHostToDevice );
		inc << < occ.numBlocks, occ.numThreads >> >( d_Data, N );
		hipMemcpy( h_Data, d_Data, size, hipMemcpyDeviceToHost );
		incData( h_Data, N );
	}

	// Get elapsed time
	hipDeviceSynchronize();
	float timeTaken = T.Elapsed();

	// Free
	free( h_Data );
	hipFree( d_Data );

	return timeTaken;
}