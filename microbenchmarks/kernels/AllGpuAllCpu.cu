#include "hip/hip_runtime.h"
#include "../util.h"
#include "microbenchmarks.h"

__global__
void inc( float * data, int N )
{
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if ( idx < N ) data[idx]++;
}

float AGACFunc::runUMA( uint32_t N, uint32_t dim, uint32_t nIt )
{
	// Create timing objects, do not start
	float timeTaken( 0 );
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );

	// Allocate data
	size_t size = sizeof( float ) * N;
	float *d_Data( 0 );
	hipMallocManaged( (void **) &d_Data, size );

	// Get max occupancy values
	LaunchParams occ = GetBestOccupancy( inc, N );

	// Start timing
	hipEventRecord( start );

	// Increment everything on CPU and GPU
	for ( int i = 0; i < nIt; i++ )
	{
		inc << < occ.numBlocks, occ.numThreads >> >( d_Data, N );
		hipDeviceSynchronize();
		for ( int j = 0; j < N; j++ )
			d_Data[j]++;
	}

	// Stop timing
	hipEventRecord( stop );
	hipEventSynchronize( stop );

	// Free
	hipFree( d_Data );

	// Get elapsed time
	hipEventElapsedTime( &timeTaken, start, stop );

	return timeTaken;
}

float AGACFunc::runHD( uint32_t N, uint32_t dim, uint32_t nIt )
{
	// Create timing objects, do not start
	float timeTaken( 0 );
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );

	// Allocate data
	size_t size = sizeof( float ) * N;
	float * h_Data( 0 ), *d_Data( 0 );
	h_Data = (float *) malloc( size );
	hipMalloc( (void **) &d_Data, size );

	// Get max occupancy values
	LaunchParams occ = GetBestOccupancy( inc, N );

	// Start timing
	hipEventRecord( start );
	
	// Increment everything on CPU and GPU
	for ( int i = 0; i < nIt; i++ )
	{
		hipMemcpy( d_Data, h_Data, size, hipMemcpyHostToDevice );
		inc << < occ.numBlocks, occ.numThreads >> >( d_Data, N );
		hipMemcpy( h_Data, d_Data, size, hipMemcpyDeviceToHost );
		for ( int j = 0; j < N; j++ )
			h_Data[j]++;
	}

	// Stop timing
	hipEventRecord( stop );
	hipEventSynchronize( stop );

	// Free
	free( h_Data );
	hipFree( d_Data );

	// Get elapsed time
	hipEventElapsedTime( &timeTaken, start, stop );

	return timeTaken;
}