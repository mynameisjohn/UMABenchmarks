#include "util.h"

int main(int argc, char ** argv){
	if (argc < 3)
		return -1;
	int N = atoi(argv[1]);
	int nIt = atoi(argv[2]);

	if (N < 0 || nIt < 0)
		return -1;

	float * h_Data(0), * d_Data(0);
   int size = sizeof(float)*N;

   h_Data = (float *)malloc(size);
   hipMalloc((void **)&d_Data, size);

   int nT(1024), nB(N/1024+1);

   for (int i=0; i<nIt; i++){
      hipMemcpy(d_Data, h_Data, size, hipMemcpyHostToDevice);
      inc<<<nB, nT>>>(d_Data, N);
      hipMemcpy(h_Data, d_Data, size, hipMemcpyDeviceToHost);
      for (int j=0; j<N; j++)
         h_Data[j]++;
   }

   free(h_Data);
   hipFree(d_Data);

	return 0;
}
