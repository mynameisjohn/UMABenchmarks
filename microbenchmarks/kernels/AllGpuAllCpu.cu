#include "hip/hip_runtime.h"
#include "util.h"
#include "microbenchmarks.h"

float AGACFunc::runUMA( uint32_t N, uint32_t dim, uint32_t nIt )
{
	// Create timing objects, do not start
	float timeTaken( 0 );
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );

	// Allocate data
	size_t size = sizeof( float ) * N;
	float *d_Data( 0 );
	hipMallocManaged( (void **) &d_Data, size );

	// Get max occupancy values
	LaunchParams occ = GetBestOccupancy( (void *)inc, N );

	// Start timing
	hipEventRecord( start );

	// Make random data between 0 and N
	makeData( d_Data, N );

	// Increment everything on CPU and GPU
	for ( int i = 0; i < nIt; i++ )
	{
		inc << < occ.numBlocks, occ.numThreads >> >( d_Data, N );
		hipDeviceSynchronize();
		incData( d_Data, N );
	}

	// Stop timing
	hipEventRecord( stop );
	hipEventSynchronize( stop );

	// Free
	hipFree( d_Data );

	// Get elapsed time
	hipEventElapsedTime( &timeTaken, start, stop );

	return timeTaken;
}

float AGACFunc::runHD( uint32_t N, uint32_t dim, uint32_t nIt )
{
	// Create timing objects, do not start
	float timeTaken( 0 );
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );

	// Allocate data
	size_t size = sizeof( float ) * N;
	float * h_Data( 0 ), *d_Data( 0 );
	h_Data = (float *) malloc( size );
	hipMalloc( (void **) &d_Data, size );

	// Get max occupancy values
	LaunchParams occ = GetBestOccupancy( (void *) inc, N );

	// Start timing
	hipEventRecord( start );

	// Make random data between 0 and N
	makeData( h_Data, N );
	
	// Increment everything on CPU and GPU
	for ( int i = 0; i < nIt; i++ )
	{
		hipMemcpy( d_Data, h_Data, size, hipMemcpyHostToDevice );
		inc << < occ.numBlocks, occ.numThreads >> >( d_Data, N );
		hipMemcpy( h_Data, d_Data, size, hipMemcpyDeviceToHost );
		incData( h_Data, N );
	}

	// Stop timing
	hipEventRecord( stop );
	hipEventSynchronize( stop );

	// Free
	free( h_Data );
	hipFree( d_Data );

	// Get elapsed time
	hipEventElapsedTime( &timeTaken, start, stop );

	return timeTaken;
}