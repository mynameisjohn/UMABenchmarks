#include "hip/hip_runtime.h"
#include "util.h"
#include "microbenchmarks.h"

__global__
void subset_G_Rand( float * in, float * out, int N, float thresh )
{
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if ( idx < N )
	{
		float val = in[idx];
		if ( val < thresh ) out[idx] = val;
	}
}

float SGACRFunc::runUMA( uint32_t N, uint32_t dim, uint32_t nIt )
{
	// Create timing objects, do not start
	float timeTaken( 0 );
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );

	// data size
	size_t size = sizeof( float ) * N;

	// separate numbers using threshold
	float thresh = (float) rand() / (float) RAND_MAX;

	// Get max occupancy values
	LaunchParams occ = GetBestOccupancy( (void *)subset_G_Rand, N );

	// Allocate and create data
	float *d_In( 0 ), *d_Out( 0 );
	hipMallocManaged( (void **) &d_In, size );
	hipMallocManaged( (void **) &d_Out, size );

	// Start timing
	hipEventRecord( start );

	// Make random input
	for ( int j = 0; j < N; j++ )
	{
		// All should be between 0 and 1
		d_In[j] = (float) rand() / (float) RAND_MAX;
		d_Out[j] = (float) rand() / (float) RAND_MAX;
	}

	// Iterate
	for ( int i = 0; i < nIt; i++ )
	{
		subset_G_Rand << <occ.numBlocks, occ.numThreads >> >( d_In, d_Out, N, thresh );
		hipDeviceSynchronize();
		for ( int j = 0; j < N; j++ )
		{
			d_In[j]++;
			d_In[j]++;
		}
	}

	// Stop Timing
	hipEventRecord( stop );
	hipEventSynchronize( stop );

	// Get elapsed time
	hipEventElapsedTime( &timeTaken, start, stop );

	// Free
	hipFree( d_In );
	hipFree( d_Out );

	return timeTaken;
}

float SGACRFunc::runHD( uint32_t N, uint32_t dim, uint32_t nIt )
{
	// Create timing objects, do not start
	float timeTaken( 0 );
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );

	// data size
	size_t size = sizeof( float ) * N;

	// separate numbers using threshold
	float thresh = (float) rand() / (float) RAND_MAX;

	// Get max occupancy values
	LaunchParams occ = GetBestOccupancy( (void *)subset_G_Rand, N );

	// Allocate and create data
	float * h_In( 0 ), *h_Out( 0 ), *d_In( 0 ), *d_Out( 0 );
	h_In = (float *) malloc( size );
	h_Out = (float *) malloc( size );
	hipMalloc( (void **) &d_In, size );
	hipMalloc( (void **) &d_Out, size );

	// Start timing
	hipEventRecord( start );

	// Make random input
	for ( int j = 0; j < N; j++ )
	{
		// All should be between 0 and 1
		h_In[j] = (float) rand() / (float) RAND_MAX;
		h_Out[j] = (float) rand() / (float) RAND_MAX;
	}

	// Iterate
	for ( int i = 0; i < nIt; i++ )
	{
		hipMemcpy( d_In, h_In, size, hipMemcpyHostToDevice );
		hipMemcpy( d_Out, h_Out, size, hipMemcpyHostToDevice );
		subset_G_Rand << <occ.numBlocks, occ.numThreads >> >( d_In, d_Out, N, thresh );
		hipMemcpy( h_In, d_In, size, hipMemcpyDeviceToHost );
		hipMemcpy( h_Out, d_Out, size, hipMemcpyDeviceToHost );
		for ( int j = 0; j < N; j++ )
		{
			h_In[j]++;
			h_Out[j]++;
		}
	}

	// Stop Timing
	hipEventRecord( stop );
	hipEventSynchronize( stop );

	// Get elapsed time
	hipEventElapsedTime( &timeTaken, start, stop );

	// Free
	free( h_In );
	free( h_Out );
	hipFree( d_In );
	hipFree( d_Out );

	return timeTaken;
}
