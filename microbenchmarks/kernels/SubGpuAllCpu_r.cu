#include "hip/hip_runtime.h"
#include "util.h"
#include "microbenchmarks.h"

__global__
void subset_G_Rand( float * in, float * out, int N, float thresh )
{
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if ( idx < N )
	{
		float val = in[idx];
		if ( val < thresh ) out[idx] = val;
	}
}

float SGACRFunc::runUMA( uint32_t N, uint32_t dim, uint32_t nIt )
{
	// Create timing objects, do not start
	float timeTaken( 0 );
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );

	// data size
	size_t size = sizeof( float ) * N;

	// separate numbers using threshold between 0 and 1
	float thresh = (float) rand() / (float) RAND_MAX;

	// Get max occupancy values
	LaunchParams occ = GetBestOccupancy( (void *)subset_G_Rand, N );

	// Allocate and create data
	float *d_In( 0 ), *d_Out( 0 );
	hipMallocManaged( (void **) &d_In, size );
	hipMallocManaged( (void **) &d_Out, size );

	// Start timing
	hipEventRecord( start );

	// Make random input between 0 and 1
	makeData( d_In, N, true );

	// Iterate
	for ( int i = 0; i < nIt; i++ )
	{
		// Run kernel
		subset_G_Rand << <occ.numBlocks, occ.numThreads >> >( d_In, d_Out, N, thresh );

		// Sync
		hipDeviceSynchronize();

		// Recreate random input, touching each input and output
		makeData( d_In, N, true );
		touchData( d_Out, N);

		// Recreate thresh
		thresh = (float) rand() / (float) RAND_MAX;
	}

	// Stop Timing
	hipEventRecord( stop );
	hipEventSynchronize( stop );

	// Get elapsed time
	hipEventElapsedTime( &timeTaken, start, stop );

	// Free
	hipFree( d_In );
	hipFree( d_Out );

	return timeTaken;
}

float SGACRFunc::runHD( uint32_t N, uint32_t dim, uint32_t nIt )
{
	// Create timing objects, do not start
	float timeTaken( 0 );
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );

	// data size
	size_t size = sizeof( float ) * N;

	// separate numbers using threshold
	float thresh = (float) rand() / (float) RAND_MAX;

	// Get max occupancy values
	LaunchParams occ = GetBestOccupancy( subset_G_Rand, N );

	// Allocate and create data
	float * h_In( 0 ), *h_Out( 0 ), *d_In( 0 ), *d_Out( 0 );
	h_In = (float *) malloc( size );
	h_Out = (float *) malloc( size );
	hipMalloc( (void **) &d_In, size );
	hipMalloc( (void **) &d_Out, size );

	// Start timing
	hipEventRecord( start );

	// Make random input between 0 and 1
	makeData( h_In, N, true );

	// Iterate
	for ( int i = 0; i < nIt; i++ )
	{
		hipMemcpy( d_In, h_In, size, hipMemcpyHostToDevice );
		hipMemcpy( d_Out, h_Out, size, hipMemcpyHostToDevice );
		subset_G_Rand << <occ.numBlocks, occ.numThreads >> >( d_In, d_Out, N, thresh );
		hipMemcpy( h_In, d_In, size, hipMemcpyDeviceToHost );
		hipMemcpy( h_Out, d_Out, size, hipMemcpyDeviceToHost );

		// Recreate random input between 0 and 1, touch output
		makeData( h_In, N, true );
		touchData( h_Out, N );

		// Reset thresh
		thresh = (float) rand() / (float) RAND_MAX;
	}

	// Stop Timing
	hipEventRecord( stop );
	hipEventSynchronize( stop );

	// Get elapsed time
	hipEventElapsedTime( &timeTaken, start, stop );

	// Free
	free( h_In );
	free( h_Out );
	hipFree( d_In );
	hipFree( d_Out );

	return timeTaken;
}