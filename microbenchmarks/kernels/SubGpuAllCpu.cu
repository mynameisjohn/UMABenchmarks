#include "hip/hip_runtime.h"
#include "util.h"

__global__
void subset_G(float * data, int N, int3 subset){
   int idx = threadIdx.x+blockDim.x*blockIdx.x;
   if (idx < N && contains(subset, idx)) data[idx]++;
}

int main(int argc, char ** argv){
	if (argc < 3)
		return -1;

	int N = atoi(argv[1]);
	int nIt = atoi(argv[2]);
   int size = sizeof(float)*N;
   int nT(1024), nB(N/1024+1);
	if (N < 0 ||nIt < 0)
		return -2;

   srand(1);

   int3 subset;
   subset.x = (int)(((float)rand()/(float)RAND_MAX) * N);
   subset.y = (int)(((float)rand()/(float)RAND_MAX) * N);
   subset.z = (int)(((float)rand()/(float)RAND_MAX) * N);

#ifdef UMA
	float * data(0);
   hipMallocManaged((void **)&data, size);

   for (int i=0; i<nIt; i++){
      subset_G<<<nB, nT>>>(data, N, subset);
		hipDeviceSynchronize();
      for (int j=0; j<N; j++)
         data[j]++;
   }

   free(data);
#else
	float * h_Data(0), * d_Data(0);
   h_Data = (float *)malloc(size);
   hipMalloc((void **)&d_Data, size);

   for (int i=0; i<nIt; i++){
      hipMemcpy(d_Data, h_Data, size, hipMemcpyHostToDevice);
      subset_G<<<nB, nT>>>(d_Data, N, subset);
      hipMemcpy(h_Data, d_Data, size, hipMemcpyDeviceToHost);
      for (int j=0; j<N; j++)
         h_Data[j]++;
   }

   free(h_Data);
   hipFree(d_Data);
#endif
	return 0;
}
