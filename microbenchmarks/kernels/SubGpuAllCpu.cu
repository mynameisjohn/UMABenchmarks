#include "hip/hip_runtime.h"
#include "../util.h"
#include "microbenchmarks.h"

__global__
void subset_G( float * data, int N, int3 subset )
{
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if ( idx < N && contains( subset, idx ) ) data[idx]++;
}

float SGACFunc::runUMA( uint32_t N, uint32_t dim, uint32_t nIt )
{
	// Create timing objects, do not start
	float timeTaken( 0 );
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );

	// Determine random subset
	int3 subset;
	subset.x = (int) ( ( (float) rand() / (float) RAND_MAX ) * N );
	subset.y = (int) ( ( (float) rand() / (float) RAND_MAX ) * N );
	subset.z = (int) ( ( (float) rand() / (float) RAND_MAX ) * N );

	// Allocate data
	size_t size = sizeof( float )*N;
	float *d_Data( 0 );
	hipMallocManaged( (void **) &d_Data, size );

	// Set input to zero
	memset( d_Data, 0, size );

	// Get max occupancy values
	LaunchParams occ = GetBestOccupancy( subset_G, N );

	// Start timing
	hipEventRecord( start );

	// Copy to device and back, then touch everything on host
	for ( int i = 0; i<nIt; i++ )
	{
		subset_G << < occ.numBlocks, occ.numThreads >> >( d_Data, N, subset );
		hipDeviceSynchronize();
		for ( int j = 0; j<N; j++ )
			d_Data[j]++;
	}

	// Free
	hipFree( d_Data );

	// Stop timing
	hipEventRecord( stop );
	hipEventSynchronize( stop );

	// Get elapsed time
	hipEventElapsedTime( &timeTaken, start, stop );

	return timeTaken;
}

float SGACFunc::runHD( uint32_t N, uint32_t dim, uint32_t nIt )
{
	// Create timing objects, do not start
	float timeTaken( 0 );
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );

	// Determine random subset
	int3 subset;
	subset.x = (int) ( ( (float) rand() / (float) RAND_MAX ) * N );
	subset.y = (int) ( ( (float) rand() / (float) RAND_MAX ) * N );
	subset.z = (int) ( ( (float) rand() / (float) RAND_MAX ) * N );

	// Allocate data
	size_t size = sizeof( float )*N;
	float * h_Data( 0 ), *d_Data( 0 );
	h_Data = (float *) malloc( size );
	hipMalloc( (void **) &d_Data, size );

	// Get max occupancy values
	LaunchParams occ = GetBestOccupancy( subset_G, N );

	// Set input to zero
	memset( h_Data, 0, size );

	// Start timing
	hipEventRecord( start );

	// Copy to device and back, then touch everything on host
	for ( int i = 0; i<nIt; i++ )
	{
		hipMemcpy( d_Data, h_Data, size, hipMemcpyHostToDevice );
		subset_G << <occ.numBlocks, occ.numThreads >> >( d_Data, N, subset );
		hipMemcpy( h_Data, d_Data, size, hipMemcpyDeviceToHost );
		for ( int j = 0; j<N; j++ )
			h_Data[j]++;
	}

	// Free
	free( h_Data );
	hipFree( d_Data );

	// Stop timing
	hipEventRecord( stop );
	hipEventSynchronize( stop );

	// Get elapsed time
	hipEventElapsedTime( &timeTaken, start, stop );

	return timeTaken;
}