#include "hip/hip_runtime.h"
#include "util.h"
#include "microbenchmarks.h"

__global__
void subset_G( float * data, int N, int3 subset )
{
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if ( idx < N && contains( subset, idx ) ) data[idx]++;
}

float SGACFunc::runUMA( uint32_t N, uint32_t dim, uint32_t nIt )
{
	CpuTimer T;

	// Determine random subset
	int3 subset = getRandomSubset( N );

	// Allocate data
	size_t size = sizeof( float )*N;
	float *d_Data( 0 );
	hipMallocManaged( (void **) &d_Data, size );

	// Get max occupancy values
	LaunchParams occ = GetBestOccupancy( (void *) subset_G, N );

	// Start timing
	T.Start();

	// Make random input between 0 and N
	makeData( d_Data, N );

	// Copy to device and back, then touch everything on host
	for ( int i = 0; i<nIt; i++ )
	{
		subset_G << < occ.numBlocks, occ.numThreads >> >( d_Data, N, subset );
		hipDeviceSynchronize();
		touchData( d_Data, N );

		// reset subset
		subset = getRandomSubset( N );
	}

	// Get elapsed time
	hipDeviceSynchronize();
	float timeTaken = T.Elapsed();

	// Free
	hipFree( d_Data );

	return timeTaken;
}

float SGACFunc::runHD( uint32_t N, uint32_t dim, uint32_t nIt )
{
	CpuTimer T;

	// Determine random subset
	int3 subset = getRandomSubset( N );

	// Allocate data
	size_t size = sizeof( float )*N;
	float * h_Data( 0 ), *d_Data( 0 );
	h_Data = (float *) malloc( size );
	hipMalloc( (void **) &d_Data, size );

	// Get max occupancy values
	LaunchParams occ = GetBestOccupancy( (void *) subset_G, N );

	// Start timing
	T.Start();

	// Create random data between 0 and N
	makeData( h_Data, N );

	// Copy to device and back, then touch everything on host
	for ( int i = 0; i<nIt; i++ )
	{
		hipMemcpy( d_Data, h_Data, size, hipMemcpyHostToDevice );
		subset_G << <occ.numBlocks, occ.numThreads >> >( d_Data, N, subset );
		hipMemcpy( h_Data, d_Data, size, hipMemcpyDeviceToHost );
		touchData( h_Data, N );

		// reset subset
		subset = getRandomSubset( N );
	}

	// Get elapsed time
	hipDeviceSynchronize();
	float timeTaken = T.Elapsed();

	// Free
	free( h_Data );
	hipFree( d_Data );

	return timeTaken;
}