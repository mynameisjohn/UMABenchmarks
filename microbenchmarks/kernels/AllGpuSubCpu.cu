#include "hip/hip_runtime.h"
#include "util.h"

int main(int argc, char ** argv){
	if (argc < 3)
		return -1;

	int N = atoi(argv[1]);
	int nIt = atoi(argv[2]);
   int nT(1024), nB(N/1024+1);

	if (N < 0 || nIt < 0)
		return -2;

   srand(1);

   int3 subset;
   subset.x = (int)(((float)rand()/(float)RAND_MAX) * N);
   subset.y = (int)(((float)rand()/(float)RAND_MAX) * N);
   subset.z = (int)(((float)rand()/(float)RAND_MAX) * N);

#ifdef UMA
   float * data(0);
   int size = sizeof(float)*N;
	hipMallocManaged((void **)&data, size);
   for (int i=0; i<nIt; i++){
      inc<<<nB, nT>>>(data, N);
		hipDeviceSynchronize();      
      for (int j=0; j<N; j++)
         if (contains(subset, j))
            data[j]++;
   }

   free(data);
#else
   float * h_Data(0), * d_Data(0);
   int size = sizeof(float)*N;
   h_Data = (float *)malloc(size);
   hipMalloc((void **)&d_Data, size);
   for (int i=0; i<nIt; i++){
      hipMemcpy(d_Data, h_Data, size, hipMemcpyHostToDevice);
      inc<<<nB, nT>>>(d_Data, N);
      hipMemcpy(h_Data, d_Data, size, hipMemcpyDeviceToHost);
      for (int j=0; j<N; j++)
         if (contains(subset, j))
            h_Data[j]++;
   }

   free(h_Data);
   hipFree(d_Data);
#endif

	return 0;
}
