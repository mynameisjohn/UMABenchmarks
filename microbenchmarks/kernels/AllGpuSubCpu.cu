#include "hip/hip_runtime.h"
#include <stdlib.h>

#include "util.h"
#include "microbenchmarks.h"


float AGSCFunc::runUMA( uint32_t N, uint32_t dim, uint32_t nIt )
{
	// Create timing objects, do not start
	float timeTaken( 0 );
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );

	// Create random subset
	int3 subset = getRandomSubset( N );

	// Allocate data
	size_t size = sizeof( float ) * N;
	float *d_Data( 0 );
	hipMallocManaged( (void **) &d_Data, size );

	// Get max occupancy values
	LaunchParams occ = GetBestOccupancy( (void *)inc, N );

	// Start timing
	hipEventRecord( start );

	// Make random data between 0 and N
	makeData( d_Data, N );

	// Run kernel, copy back to host, only touch subset on CPU
	for ( int i = 0; i < nIt; i++ )
	{
		inc << < occ.numBlocks, occ.numThreads >> >( d_Data, N );
		hipDeviceSynchronize();

		touchSubset( d_Data, N, subset );

		// reset subset
		subset = getRandomSubset( N );
	}

	// Stop timing
	hipEventRecord( stop );
	hipEventSynchronize( stop );

	// Free
	hipFree( d_Data );

	// Get elapsed time
	hipEventElapsedTime( &timeTaken, start, stop );

	return timeTaken;
}

float AGSCFunc::runHD( uint32_t N, uint32_t dim, uint32_t nIt )
{
	// Create timing objects, do not start
	float timeTaken( 0 );
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );

	// Create random subset
	int3 subset = getRandomSubset( N );

	// Allocate data
	size_t size = sizeof( float ) * N;
	float * h_Data( 0 ), *d_Data( 0 );
	h_Data = (float *) malloc( size );
	hipMalloc( (void **) &d_Data, size );

	// Get max occupancy values
	LaunchParams occ = GetBestOccupancy( (void *) inc, N );

	// Start timing
	hipEventRecord( start );

	// Make random input between 0 and N
	makeData( h_Data, N );

	// Run kernel, copy back to host, only touch subset on CPU
	for ( int i = 0; i < nIt; i++ )
	{
		hipMemcpy( d_Data, h_Data, size, hipMemcpyHostToDevice );
		inc << < occ.numBlocks, occ.numThreads >> >( d_Data, N );
		hipMemcpy( h_Data, d_Data, size, hipMemcpyDeviceToHost );

		touchSubset( h_Data, N, subset );

		// reset subset
		subset = getRandomSubset( N );
	}

	// Stop timing
	hipEventRecord( stop );
	hipEventSynchronize( stop );

	// Free
	free( h_Data );
	hipFree( d_Data );

	// Get elapsed time
	hipEventElapsedTime( &timeTaken, start, stop );

	return timeTaken;
}